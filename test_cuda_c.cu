
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void test(){
    printf("Hello from thread %d at block %d!\n", threadIdx.x, blockIdx.x);
}

void say_hello_cuda(){
    // your code for initialization, copying data to device memory,
    test<<<2,2>>>(); //kernel call
    hipDeviceSynchronize();
    //your code for copying back the result to host memory & return
 }